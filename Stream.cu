
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <sys/time.h>

///////////////////////////////////////////////////////////////////////////////
// Kernel
///////////////////////////////////////////////////////////////////////////////

__global__ void gpuKernel( double *d_C, double *d_A, double *d_B, int DATA_N ) {

  for(int pos=(blockIdx.x*blockDim.x)+threadIdx.x;
          pos < DATA_N;
          pos += blockDim.x*gridDim.x) {
     d_C[pos] = d_A[pos] + d_B[pos];
  }
}

////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
double RandFloat(double low, double high){
  double t = (double)rand() / (double)RAND_MAX;
  return (1.0f - t) * low + t * high;
}


#define GPU_ERROR(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    printf( "GPUassert: %s in %s:%d\n", hipGetErrorString(code), file, line);

    if (abort)
      exit(code);
  }
}

double dtime() {
  double tseconds = 0;
  struct timeval t;
  gettimeofday(&t, NULL);
  tseconds = (double)t.tv_sec + (double)t.tv_usec * 1.0e-6;
  return tseconds;
}
///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
  double *h_A, *h_B, *h_C_CPU, *h_C_GPU;
  double *d_A, *d_B, *d_C;
  double  ws1,we1,ws0,we0;

  int i,blocks=8,threads=8;

  //Total number of data elements
  int    DATA_N =  1000;

  if(argc>1) DATA_N=atoi(argv[1]);
  if(argc>2) blocks=atoi(argv[2]);
  if(argc>3) threads=atoi(argv[3]);

  const int   DATA_SZ = DATA_N * sizeof(double);
  const int RESULT_SZ = DATA_N  * sizeof(double);


  printf("Initializing data...\n");
  printf("...allocating CPU memory.\n");
  GPU_ERROR(hipHostAlloc( (void**)&h_A, (sizeof (double) * DATA_N), 0)) ;
  GPU_ERROR(hipHostAlloc( (void**)&h_B, (sizeof (double) * DATA_N), 0)) ;
  GPU_ERROR(hipHostAlloc( (void**)&h_C_CPU, (sizeof (double) * DATA_N), 0)) ;
  GPU_ERROR(hipHostAlloc( (void**)&h_C_GPU, (sizeof (double) * DATA_N), 0)) ;

  printf("...allocating GPU memory.\n");
  GPU_ERROR( hipMalloc((void **)&d_A, DATA_SZ))   ;
  GPU_ERROR(  hipMalloc((void **)&d_B, DATA_SZ))   ;
  GPU_ERROR(hipMalloc((void **)&d_C, RESULT_SZ)) ;

  printf("...generating input data in CPU mem.\n");
  srand(123);
  //Generating input data on CPU
  for(i = 0; i < DATA_N; i++){
    h_A[i] = RandFloat(0.0f, 1.0f);
    h_B[i] = RandFloat(0.0f, 1.0f);
  }

  printf("...copying input data to GPU mem.\n");

  ws0 = dtime();

  //Copy data to GPU memory for further processing 
  GPU_ERROR(hipMemcpy(d_A, h_A, DATA_SZ, hipMemcpyHostToDevice));
  GPU_ERROR(hipMemcpy(d_B, h_B, DATA_SZ, hipMemcpyHostToDevice));
  printf("Data init done. Now executing GPU kernel ...\n");

  GPU_ERROR(hipDeviceSynchronize()) ;
  ws1 = dtime();
  GPU_ERROR(hipDeviceSynchronize()) ;

  dim3 threadsPerBlock(threads);           // (x,y) or (x,y,z) works too
  dim3 numBlocks(blocks);                // (x,y) works too
                                        // also ProbSize/threadsPerBlock.x etc.

  //Kernel Call:  <<<BLOCKS, THREADS_PER_BLOCK>>>
  gpuKernel<<<numBlocks,threadsPerBlock>>>(d_C, d_A, d_B, DATA_N);
  GPU_ERROR(hipPeekAtLastError());
  GPU_ERROR(hipDeviceSynchronize());
  we1 = dtime();


  printf("Reading back GPU result...\n");
  //Read back GPU results to compare them to CPU results
  GPU_ERROR(hipMemcpy(h_C_GPU, d_C, RESULT_SZ, hipMemcpyDeviceToHost)) ;
  we0 = dtime();



  printf("CUDA performance: RAW: %lf MIt/s     incl. transf.: %lf MIt/s\n",
	 (double)DATA_N/(we1-ws1)/1e6,
	 (double)DATA_N/(we0-ws0)/1e6);

  printf("Shutting down...\n");
  hipFree(d_C) ;
  hipFree(d_B) ;
  hipFree(d_A) ;
  hipFree(h_C_GPU);
  hipFree(h_C_CPU);
  hipFree(h_B);
  hipFree(h_A);
}
